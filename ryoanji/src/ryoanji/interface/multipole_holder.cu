#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief  Interface for calculation of multipole moments
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <thrust/device_vector.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/util/reallocate.hpp"
#include "ryoanji/nbody/cartesian_qpole.hpp"
#include "ryoanji/nbody/upwardpass.cuh"
#include "ryoanji/nbody/upsweep_cpu.hpp"
#include "ryoanji/nbody/traversal.cuh"
#include "multipole_holder.cuh"

namespace ryoanji
{

template<class Tc, class Th, class Tm, class Ta, class Tf, class KeyType, class MType>
class MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, MType>::Impl
{
public:
    Impl() {}

    void upsweep(const Tc* x, const Tc* y, const Tc* z, const Tm* m, const cstone::Octree<KeyType>& globalOctree,
                 const cstone::FocusedOctree<KeyType, Tf, cstone::GpuTag>& focusTree, const cstone::LocalIndex* layout,
                 MType* multipoles)
    {
        constexpr int                  numThreads = UpsweepConfig::numThreads;
        const cstone::Octree<KeyType>& octree     = focusTree.octree();

        TreeNodeIndex numLeaves = focusTree.octree().numLeafNodes();
        resize(numLeaves);

        auto centers       = focusTree.expansionCenters();
        auto globalCenters = focusTree.globalExpansionCenters();

        const TreeNodeIndex* leafToInternal = octree.internalOrder().data();
        memcpyH2D(leafToInternal, numLeaves, rawPtr(leafToInternal_));

        const TreeNodeIndex* internalToLeaf = octree.toLeafOrder().data();
        memcpyH2D(internalToLeaf, internalToLeaf_.size(), rawPtr(internalToLeaf_));

        const TreeNodeIndex* childOffsets = octree.childOffsets().data();
        memcpyH2D(childOffsets, childOffsets_.size(), rawPtr(childOffsets_));

        memcpyH2D(layout, layout_.size(), rawPtr(layout_));
        memcpyH2D(centers.data(), centers.size(), rawPtr(centers_));

        computeLeafMultipoles<<<(numLeaves - 1) / numThreads + 1, numThreads>>>(
            x, y, z, m, rawPtr(leafToInternal_), numLeaves, rawPtr(layout_), rawPtr(centers_), rawPtr(multipoles_));

        //! first upsweep with local data
        int  numLevels  = 21;
        auto levelRange = octree.levelRange();
        for (int level = numLevels - 1; level >= 0; level--)
        {
            int numCellsLevel = levelRange[level + 1] - levelRange[level];
            int numBlocks     = (numCellsLevel - 1) / numThreads + 1;
            upsweepMultipoles<<<numBlocks, numThreads>>>(levelRange[level], levelRange[level + 1],
                                                         rawPtr(childOffsets_), rawPtr(centers_), rawPtr(multipoles_));
        }

        memcpyD2H(rawPtr(multipoles_), multipoles_.size(), multipoles);

        auto ryUpsweep = [](auto levelRange, auto childOffsets, auto M, auto centers)
        { upsweepMultipoles(levelRange, childOffsets, centers, M); };

        gsl::span multipoleSpan{multipoles, size_t(octree.numTreeNodes())};
        cstone::globalFocusExchange(globalOctree, focusTree, multipoleSpan, ryUpsweep, globalCenters.data());

        focusTree.peerExchange(multipoleSpan, static_cast<int>(cstone::P2pTags::focusPeerCenters) + 1);

        // H2D multipoles
        memcpyH2D(multipoles, multipoles_.size(), rawPtr(multipoles_));

        //! second upsweep with leaf data from peer and global ranks in place
        for (int level = numLevels - 1; level >= 0; level--)
        {
            int numCellsLevel = levelRange[level + 1] - levelRange[level];
            int numBlocks     = (numCellsLevel - 1) / numThreads + 1;
            upsweepMultipoles<<<numBlocks, numThreads>>>(levelRange[level], levelRange[level + 1],
                                                         rawPtr(childOffsets_), rawPtr(centers_), rawPtr(multipoles_));
        }
    }

    float compute(LocalIndex firstBody, LocalIndex lastBody, const Tc* x, const Tc* y, const Tc* z, const Tm* m,
                  const Th* h, Tc G, Ta* ax, Ta* ay, Ta* az)
    {
        resetTraversalCounters<<<1, 1>>>();

        constexpr int numWarpsPerBlock = TravConfig::numThreads / cstone::GpuConfig::warpSize;

        LocalIndex numBodies = lastBody - firstBody;

        // each target gets a warp (numWarps == numTargets)
        int numWarps  = (numBodies - 1) / TravConfig::targetSize + 1;
        int numBlocks = (numWarps - 1) / numWarpsPerBlock + 1;
        numBlocks     = std::min(numBlocks, TravConfig::maxNumActiveBlocks);

        LocalIndex poolSize = TravConfig::memPerWarp * numWarpsPerBlock * numBlocks;

        reallocateGeneric(globalPool_, poolSize, 1.05);
        traverse<<<numBlocks, TravConfig::numThreads>>>(
            firstBody, lastBody, {1, 9}, x, y, z, m, h, rawPtr(childOffsets_), rawPtr(internalToLeaf_), rawPtr(layout_),
            rawPtr(centers_), rawPtr(multipoles_), G, (int*)(nullptr), ax, ay, az, rawPtr(globalPool_));
        float totalPotential;
        checkGpuErrors(hipMemcpyFromSymbol(&totalPotential, HIP_SYMBOL(totalPotentialGlob), sizeof(float)));

        return 0.5f * Tc(G) * totalPotential;
    }

    const MType* deviceMultipoles() const { return rawPtr(multipoles_); }

private:
    void resize(size_t numLeaves)
    {
        double growthRate = 1.01;
        size_t numNodes   = numLeaves + (numLeaves - 1) / 7;

        auto dealloc = [](auto& v)
        {
            v.clear();
            v.shrink_to_fit();
        };

        if (numLeaves > leafToInternal_.capacity())
        {
            dealloc(leafToInternal_);
            dealloc(internalToLeaf_);
            dealloc(childOffsets_);
            dealloc(layout_);
            dealloc(centers_);
            dealloc(multipoles_);
        }

        reallocateGeneric(leafToInternal_, numLeaves, growthRate);
        reallocateGeneric(internalToLeaf_, numNodes, growthRate);
        reallocateGeneric(childOffsets_, numNodes, growthRate);

        reallocateGeneric(layout_, numLeaves + 1, growthRate);

        reallocateGeneric(centers_, numNodes, growthRate);
        reallocateGeneric(multipoles_, numNodes, growthRate);
    }

    thrust::device_vector<TreeNodeIndex> leafToInternal_;
    thrust::device_vector<TreeNodeIndex> internalToLeaf_;
    thrust::device_vector<TreeNodeIndex> childOffsets_;

    thrust::device_vector<LocalIndex> layout_;

    thrust::device_vector<Vec4<Tf>> centers_;
    thrust::device_vector<MType>    multipoles_;

    thrust::device_vector<int> globalPool_;
};

template<class Tc, class Th, class Tm, class Ta, class Tf, class KeyType, class MType>
MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, MType>::MultipoleHolder()
    : impl_(new Impl())
{
}

template<class Tc, class Th, class Tm, class Ta, class Tf, class KeyType, class MType>
MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, MType>::~MultipoleHolder() = default;

template<class Tc, class Th, class Tm, class Ta, class Tf, class KeyType, class MType>
void MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, MType>::upsweep(
    const Tc* x, const Tc* y, const Tc* z, const Tm* m, const cstone::Octree<KeyType>& globalTree,
    const cstone::FocusedOctree<KeyType, Tf, cstone::GpuTag>& focusTree, const LocalIndex* layout, MType* multipoles)
{
    impl_->upsweep(x, y, z, m, globalTree, focusTree, layout, multipoles);
}

template<class Tc, class Th, class Tm, class Ta, class Tf, class KeyType, class MType>
float MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, MType>::compute(LocalIndex firstBody, LocalIndex lastBody,
                                                                   const Tc* x, const Tc* y, const Tc* z, const Tm* m,
                                                                   const Th* h, Tc G, Ta* ax, Ta* ay, Ta* az)
{
    return impl_->compute(firstBody, lastBody, x, y, z, m, h, G, ax, ay, az);
}

template<class Tc, class Th, class Tm, class Ta, class Tf, class KeyType, class MType>
const MType* MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, MType>::deviceMultipoles() const
{
    return impl_->deviceMultipoles();
}

#define MHOLDER_SPH(Tc, Th, Tm, Ta, Tf, KeyType, MVal)                                                                 \
    template class MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, SphericalMultipole<MVal, 4>>

MHOLDER_SPH(double, double, double, double, double, uint64_t, double);
MHOLDER_SPH(double, double, float, double, double, uint64_t, float);
MHOLDER_SPH(float, float, float, float, float, uint64_t, float);

#define MHOLDER_CART(Tc, Th, Tm, Ta, Tf, KeyType, MVal)                                                                \
    template class MultipoleHolder<Tc, Th, Tm, Ta, Tf, KeyType, CartesianQuadrupole<MVal>>

MHOLDER_CART(double, double, double, double, double, uint64_t, double);
MHOLDER_CART(double, double, float, double, double, uint64_t, float);
MHOLDER_CART(float, float, float, float, float, uint64_t, float);

} // namespace ryoanji
